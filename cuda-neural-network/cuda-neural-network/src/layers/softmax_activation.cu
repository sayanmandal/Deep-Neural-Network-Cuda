#include "hip/hip_runtime.h"
#include "softmax_activation.hh"
#include "../nn_utils/nn_exception.hh"
#include <iostream>


void printmatrix1(Matrix& m){
	for(int i = 0 ; i < 2; i++){
		for(int j = 0 ; j < m.shape.y ; j++)
			std::cout << m[j * m.shape.x + i] << " ";
		std::cout << std::endl;
	}

}


softmaxActivation::softmaxActivation(std::string name) {
  this->name = name;
}

softmaxActivation::~softmaxActivation()
{ }


__global__ void softmax_trivial(float* softmaxP, float* b, int rows, int cols){
	int tid = threadIdx.x;
	int bid = blockIdx.x;

	float _max = -100000000.0;
	float sum = 0.0;

	if(tid * cols + bid < rows * cols){
		for(int i = 0 ; i < rows ; i++)	_max = max(_max, b[i * cols + bid]);
		for(int i = 0 ; i < rows ; i++)	softmaxP[i * cols + bid] = __expf(b[i * cols + bid] - _max);
		for(int i = 0 ; i < rows ; i++)	sum += softmaxP[i * cols + bid];
		for(int i = 0 ; i < rows ; i++)	softmaxP[i * cols + bid] /= sum;
	}
}

  /*
  * blocks : cuSoftMaxP->rows
  * threads: cuSoftMaxP->cols
  * shared : sizeof(float) * cuSoftMaxP->cols * 2
  */
__global__ void g_getSoftMaxP(float* softMaxP, float* b, int cols, int row){
  int bid = blockIdx.x;
	extern __shared__ float _share[];
	float * _max = _share;
	float * _sum = _share + blockDim.x;
	float* sp = softMaxP + bid;
	_sum[threadIdx.x] = 0.0;
	_max[threadIdx.x] = -100000000.0;
	for(int tid = threadIdx.x * cols + blockIdx.x; tid < row * cols; tid += cols){
		//int id = tid + threadIdx.x;
		//if(id < cols){
			sp[tid] += b[tid];
			_max[threadIdx.x] = max(_max[threadIdx.x], sp[tid]);
		//}
	}
	__syncthreads();
	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < (len >> 1))
		{
			if(_max[threadIdx.x] < _max[threadIdx.x + skip])
			{
				_max[threadIdx.x] = _max[threadIdx.x + skip];
			}
		}
		len = (len + 1) >> 1;
	}
	__syncthreads();
	for(int tid = threadIdx.x * cols + blockIdx.x; tid < row * cols; tid += cols){
	//	int id = tid + threadIdx.x;
		//if(id < cols){
			sp[tid] -= _max[0];
			sp[tid] = __expf(sp[tid]);
			_sum[threadIdx.x] += sp[tid];
		//}
	}
	__syncthreads();
	len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < (len >> 1))
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = (len + 1) >> 1;
	}
	__syncthreads();
	for(int tid = threadIdx.x * cols + blockIdx.x; tid < row * cols; tid += cols){
		//int id = tid + threadIdx.x;
		//if(id < cols){
			sp[tid] /= _sum[0];
		//}
	}
}



Matrix& softmaxActivation::forward(Matrix& Z) {
	this->Z = Z;
	A.allocateMemoryIfNotAllocated(Z.shape);
  //int szy = Z.shape.y;
	/*
  dim3 block  = A.shape.x;
  //dim3 thread = std::min(512, szy);
  //convert
  //Z.copyDeviceToHost();
  //printmatrix1(Z);

  int say = A.shape.y;
  int threads = std::min(512, say);
  g_getSoftMaxP<<<block, threads, sizeof(float) * threads * 2>>>(
  A.data_device.get(),
  Z.data_device.get(),
  A.shape.x, A.shape.y);
	*/

	dim3 block = A.shape.x;
	dim3 threads = 1;
	softmax_trivial<<<block, threads>>>(A.data_device.get(), Z.data_device.get(), A.shape.y, A.shape.x);


  hipStreamSynchronize(0);
  //A.copyDeviceToHost();
  //printmatrix1(A);
  /*
  std::cout << Z.shape.x << " " << Z.shape.y << std::endl;
  Z.copyDeviceToHost();
  for(int i = 0 ; i < Z.shape.x ; i++){
    for(int j = 0 ; j < Z.shape.y ; j++){
      std::cout << Z.data_host.get()[i * Z.shape.y + j] << " ";
    }
    std::cout << std::endl;
  }
  */

  //std::cout << A.shape.x << " " << A.shape.y << std::endl;
  //getLastCudaError("g_getSoftMaxP");

	NNException::throwIfDeviceErrorsOccurred("Cannot perform softmax forward propagation.");

	return A;
}


__global__ void softmaxActivationBackprop(float* Z, float* dA, float* dZ,
										  int Z_x_dim, int Z_y_dim){

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if(index < Z_x_dim * Z_y_dim){
    dZ[index] = dA[index];
  }

}


Matrix& softmaxActivation::backprop(Matrix& dA, float learning_rate) {
  dZ.allocateMemoryIfNotAllocated(Z.shape);
  /*
  dA.copyDeviceToHost();
  for(int i = 0 ; i < dA.shape.x ; i++){
    for(int j = 0 ; j < dA.shape.y ; j++){
      std::cout << dA.data_host.get()[i * dA.shape.y + j] << " ";
    }
    std::cout << std::endl;
  }
  */

	dim3 block_size(256);
	dim3 num_of_blocks((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);
	softmaxActivationBackprop<<<num_of_blocks, block_size>>>(Z.data_device.get(), dA.data_device.get(),
															 dZ.data_device.get(),
															 Z.shape.x, Z.shape.y);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform softmax back propagation");

	return dZ;

}
